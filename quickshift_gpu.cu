#include "hip/hip_runtime.h"
#include "quickshift_cmn.h"

texture<float, 3, hipReadModeElementType> texture_pixels;
texture<float, 2, hipReadModeElementType> texture_density;

__device__ float get_pixel(int with_texture, int x, int y, int ch, int height, int width, const float * data){
	if(with_texture) return tex3D(texture_pixels, x+0.5f, y+0.5f, ch+0.5f);
	else return data[x + height*y + width*height*ch];
}

__device__ float get_density(int with_texture, int x, int y, int height, float * E){
	if(with_texture) return tex2D(texture_density, x+0.5f, y+0.5f);
	else return E[x + height*y];
}

__device__ float distance(const float * data, int height, int width, int channels, float * v, int x_col, int x_row, int y_col, int y_row, int with_texture){
	int d1 = y_col - x_col;
	int d2 = y_row - x_row;
	int k;
	float dist = d1*d1 + d2*d2;										 
	for (k = 0; k < channels; ++k) {
		float d = v[k] - get_pixel(with_texture,y_col,y_row,k,height,width,data);
		dist += d*d;
	}
	return dist;
}

int divide_grid(int num, int den){
	return (num % den != 0) ? (num / den + 1) : (num / den);
}


__global__ void find_neighbors(const float * data, int height, int width, int channels, float * E, float dist, int Rd, float * map, float * gaps, int with_texture){	 
	
	// thread index
	int x_col = blockIdx.y * blockDim.y + threadIdx.y;
	int x_row = blockIdx.x * blockDim.x + threadIdx.x;
	if (x_col >= height || x_row >= width) return; // out of bounds

	// varibales for best neighbor
	int y_col,y_row;
	float E0 = get_density(with_texture,x_col,x_row,height,E);
	float d_best = INF;
	float y_col_best = x_col;
	float y_row_best = x_row; 

	// initialize boundaries from dist
	int y_col_min = MAX(x_col - Rd, 0);
	int y_col_max = MIN(x_col + Rd, height-1);
	int y_row_min = MAX(x_row - Rd, 0);
	int y_row_max = MIN(x_row + Rd, width-1);
 
	// cache the center value
	float v[3];
	for (int k = 0; k < channels; ++k)
		v[k] = get_pixel(with_texture,x_col,x_row,k,height,width,data);

	for (y_row = y_row_min; y_row <= y_row_max; ++ y_row) {
		for (y_col = y_col_min; y_col <= y_col_max; ++ y_col) {
			if (get_density(with_texture,y_col,y_row,height,E) > E0) {
				float Dij = distance(data,height,width,channels,v,x_col,x_row,y_col,y_row,with_texture);
				if (Dij <= dist*dist && Dij < d_best) {
					d_best = Dij;
					y_col_best = y_col;
					y_row_best = y_row;
				}
			}
		}
	}

	// map is the index of the best pair
	// gaps is the minimal distance, INF = root
	map [x_col + height * x_row] = y_col_best + height * y_row_best;
	if (map[x_col + height * x_row] != x_col + height * x_row) gaps[x_col + height * x_row] = sqrt(d_best);
	else gaps[x_col + height * x_row] = d_best;
}

__global__ void compute_density(const float * data, int height, int width, int channels, int R, float sigma, float * E, int with_texture){

	// thread index
	int x_col = blockIdx.y * blockDim.y + threadIdx.y;
	int x_row = blockIdx.x * blockDim.x + threadIdx.x;
	if (x_col >= height || x_row >= width) return; // out of bounds

	// initialize boundaries from sigma
	int y_col,y_row;
	int y_col_min = MAX(x_col - R, 0);
	int y_col_max = MIN(x_col + R, height-1);
	int y_row_min = MAX(x_row - R, 0);
	int y_row_max = MIN(x_row + R, width-1);
	float Ei = 0;

	// cache the center value in registers
	float v[3];
	for (int k = 0; k < channels; ++k)
		v[k] = get_pixel(with_texture,x_col,x_row,k,height,width,data);

	// for each pixel in the area (sigma) compute the distance between it and the source pixel
	for (y_row = y_row_min; y_row <= y_row_max; ++ y_row) {
		for (y_col = y_col_min; y_col <= y_col_max; ++ y_col) {
			float Dij = distance(data,height,width,channels,v,x_col,x_row,y_col,y_row,with_texture);
			float Fij = exp(-Dij / (2*sigma*sigma));
			Ei += Fij;
		}
	}
	// normalize
	E[x_col + height * x_row] = Ei / ((y_col_max-y_col_min)*(y_row_max-y_row_min));
}


void quickshift_gpu(qs_image image, float sigma, float dist, float * map, float * gaps, float * E, int with_texture){

	hipArray * cuda_array_pixels;
	hipArray * cuda_array_density;

	// texture for the image
	if(with_texture){

		hipChannelFormatDesc descr_pixels = hipCreateChannelDesc<float>();

		texture_pixels.normalized = false;
		texture_pixels.filterMode = hipFilterModePoint;

		hipExtent const ext = {image.height, image.width, image.channels};
		hipMalloc3DArray(&cuda_array_pixels, &descr_pixels, ext);

		hipMemcpy3DParms copyParams = {0};
		copyParams.extent = make_hipExtent(image.height, image.width, image.channels);
		copyParams.kind = hipMemcpyHostToDevice;
		copyParams.dstArray = cuda_array_pixels;
		copyParams.srcPtr = make_hipPitchedPtr((void*)&image.data[0], ext.width*sizeof(float), ext.width, ext.height);
		hipMemcpy3D(&copyParams);

		hipBindTextureToArray(texture_pixels, cuda_array_pixels, descr_pixels);

	}

	// variables
	float *map_cuda, *E_cuda, *gaps_cuda, *data;
	int height = image.height;
	int width = image.width;
	int channels = image.channels;
	int R = (int) ceil (3 * sigma);
	int Rd = (int) ceil (dist);

	// allocate memory on device
	unsigned int size = image.height*image.width * sizeof(float);
	hipMalloc((void**) &data, size*image.channels);
	hipMalloc((void**) &map_cuda, size);
	hipMalloc((void**) &gaps_cuda, size);
	hipMalloc((void**) &E_cuda, size);

	hipMemcpy(data, image.data, size*image.channels, hipMemcpyHostToDevice);
	hipMemset(E_cuda, 0, size);

	// compute density (and copy result to host)
	dim3 dimBlock(32,4,1);
	dim3 dimGrid(divide_grid(width, dimBlock.x), divide_grid(height, dimBlock.y), 1);
	compute_density <<<dimGrid,dimBlock>>> (data, height, width, channels, R, sigma, E_cuda,with_texture);
	hipDeviceSynchronize();
	hipMemcpy(E, E_cuda, size, hipMemcpyDeviceToHost);

	// texture for density
	if(with_texture){

		hipChannelFormatDesc descr_density = hipCreateChannelDesc<float>();

		texture_density.normalized = false;
		texture_density.filterMode = hipFilterModePoint;

		hipMallocArray(&cuda_array_density, &descr_density, image.height, image.width);
		hipMemcpyToArray(cuda_array_density, 0, 0, E, sizeof(float)*image.height*image.width, hipMemcpyHostToDevice);

		hipBindTextureToArray(texture_density, cuda_array_density, descr_density);

		hipDeviceSynchronize();
	}

	// find neighbors (and copy result to host)
	find_neighbors <<<dimGrid,dimBlock>>> (data, height ,width, channels, E_cuda, dist, Rd, map_cuda, gaps_cuda, with_texture);
	hipDeviceSynchronize();
	hipMemcpy(map, map_cuda, size, hipMemcpyDeviceToHost);
	hipMemcpy(gaps, gaps_cuda, size, hipMemcpyDeviceToHost);

	// cleanup
	hipFree(data);
	hipFree(map_cuda);
	hipFree(gaps_cuda);
	hipFree(E_cuda);
	if(with_texture){
		hipUnbindTexture(texture_pixels);
		hipFreeArray(cuda_array_pixels);
		hipUnbindTexture(texture_density);
		hipFreeArray(cuda_array_density);
	}
}
